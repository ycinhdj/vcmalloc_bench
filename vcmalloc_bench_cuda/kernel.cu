#include "hip/hip_runtime.h"
﻿
#include "hip/hip_runtime.h"
#include ""
#include <iostream>
#include "vcmalloc.h"
#include <mimalloc/include/mimalloc.h>
#include "json/json.h"
#include <string>
#include <pcm/src/cpucounters.h>
#include <psapi.h>
//#include <windows.h>

using namespace std;
using namespace pcm;
using namespace vcma;


__global__ void vecAdd(double* a, double* b, double* c)
{

    int id = blockIdx.x;
    c[id] = a[id] + b[id];
}

__global__ void vecAdd_compact(double* data)
{
    double* a = data;
    double* b = data + gridDim.x;
    double* c = data + gridDim.x*2;

    int id = blockIdx.x;
    c[id] = a[id] + b[id];
}

void inline cuda_check() {
    hipError_t err = hipGetLastError();
    if (err != hipSuccess) {
        printf("Error: %s\n", hipGetErrorString(err));
        throw std::string("We have a problem");
    }
}


void to_json(Json::Value n_rslt, int n_test) {
    Json::Value rslt;

    std::fstream rslt_file;
    rslt_file.open("results.json", std::ios::in);

    //Read current json file
    Json::CharReaderBuilder rbuilder;
    rbuilder["collectComments"] = false;
    std::string errs;
    bool ok = Json::parseFromStream(rbuilder, rslt_file, &rslt, &errs);
    rslt_file.close();

    //Add new data

    string scenario = n_rslt["scenario"].asString();
    string allocator = n_rslt["allocator"].asString();
    string datatype = n_rslt["datatype"].asString();
    string size = n_rslt["size"].asString();

    auto operations = n_rslt["operations"].getMemberNames();

    for (auto operation : operations) {
        auto results = n_rslt["operations"][operation].getMemberNames();
        for (auto result : results) {
            rslt[scenario][allocator][datatype][size][operation][result][to_string(n_test)] = n_rslt["operations"][operation][result].asString();
        }
    }

    //Write file
    rslt_file.open("results.json", std::ios::out);
    rslt_file << rslt.toStyledString() << std::endl;
    rslt_file.close();

    cout << scenario << " " << allocator << " " << datatype << " " << size << " " << n_test << endl;
}

void result_filler(
    Json::Value& result,
    std::string operation,
    clock_t c_before, clock_t c_after,
    SystemCounterState pcm_before, SystemCounterState pcm_after,
    PROCESS_MEMORY_COUNTERS ps_before, PROCESS_MEMORY_COUNTERS ps_after,
    hipEvent_t cu_before, hipEvent_t cu_after) {

    result["operations"][operation]["Memory Consumption"] = ps_after.PagefileUsage - ps_before.PagefileUsage;

    result["operations"][operation]["Cycles"] = (int)(c_after - c_before);

    //result["operations"][operation]["PCM Cycles"]				=	getCycles(pcm_before, pcm_after);
    //result["operations"][operation]["IPC"]						=	getIPC(pcm_before, pcm_after);
    result["operations"][operation]["L2 Cache Misses"] = getL2CacheMisses(pcm_before, pcm_after);
    //result["operations"][operation]["L2 Cache Hits"]			=	getL2CacheHits(pcm_before, pcm_after);
    //result["operations"][operation]["L2 Cache Hit Ratio"]		=	getL2CacheHitRatio(pcm_before, pcm_after);
    result["operations"][operation]["L3 Cache Misses"] = getL3CacheMisses(pcm_before, pcm_after);
    //result["operations"][operation]["L3 Cache Hits"]			=	getL3CacheHits(pcm_before, pcm_after);
    //result["operations"][operation]["L3 Cache Hit Ratio"]		=	getL3CacheHitRatio(pcm_before, pcm_after);
    result["operations"][operation]["Consumed Joules"] = getConsumedJoules(pcm_before, pcm_after);
    //result["operations"][operation]["DRAM Consumed Joules"]		=	getDRAMConsumedJoules(pcm_before, pcm_after);

    float time;
    hipEventElapsedTime(&time, cu_before, cu_after);
    result["operations"][operation]["CUDA Time"] = time;


}


Json::Value vcmalloc_cuda_copy(SIZE_T param_num) {

    SIZE_T allocSize = pow(2, 27);
    SIZE_T N = sqrt((allocSize / param_num)) / sqrt(sizeof(double));
    SIZE_T M = ((allocSize / param_num) / (N*sizeof(double)));

    Json::Value result;
    result["allocator"] = "vcm";
    result["scenario"] = "CUDA";
    result["datatype"] = typeid(double).name();
    result["size"] = param_num;


    
    //evaluation variables
    clock_t c_before, c_after;						//clock
    SystemCounterState pcm_before, pcm_after;		//intel pcm
    PROCESS_MEMORY_COUNTERS ps_before, ps_after;	//windows process status
    hipEvent_t cu_before, cu_after;
    hipEventCreate(&cu_before);
    hipEventCreate(&cu_after);

    
    double **h_data;
    double *d_data;

    void* vc = vccalloc((SIZE_T)1024 * 1024 * 1024 * 1024);
    vcmalloc(vc, h_data, param_num);
    


    //Host allocation
    for (size_t i = 0; i < param_num; i++)
        vcmalloc(vc, h_data[i], N * M);
    
    // Host initialization
    for (SIZE_T i = 0; i < param_num; i++)
        for (size_t j = 0; j < N * M; j++)
            h_data[i][j] = i * N * M + j;

    // Device allocation
    hipMalloc((void**)&d_data, sizeof(double) * N * M * param_num);
    cuda_check();

    // Transfer data from host to device memory
    //before
    GetProcessMemoryInfo(GetCurrentProcess(), &ps_before, sizeof(ps_before));
    pcm_before = getSystemCounterState();
    c_before = clock();
    hipEventRecord(cu_before, 0);

    hipMemcpy(d_data, *h_data, sizeof(double) * N * M * param_num, hipMemcpyHostToDevice);
    cuda_check();

    //after
    hipEventRecord(cu_after, 0);
    hipEventSynchronize(cu_after);
    c_after = clock();
    pcm_after = getSystemCounterState();
    GetProcessMemoryInfo(GetCurrentProcess(), &ps_after, sizeof(ps_after));
    result_filler(result, "Host to Device", c_before, c_after, pcm_before, pcm_after, ps_before, ps_after, cu_before, cu_after);
    
    // Launch the calculation
    // vecAdd<<<M*N, 1>>> (&d_data[0], &d_data[N*M], &d_data[N*M*2]);
    // vecAdd_compact<<<M*N, 1>>> (d_data);
    //hipDeviceSynchronize();

    //Transfer data from device to host
    //before
    GetProcessMemoryInfo(GetCurrentProcess(), &ps_before, sizeof(ps_before));
    pcm_before = getSystemCounterState();
    c_before = clock();
    hipEventRecord(cu_before, 0);

    hipMemcpy(h_data[param_num-1], &d_data[M*N*(param_num-1)], sizeof(double) * N * M, hipMemcpyDeviceToHost);
    cuda_check();

    //after
    hipEventRecord(cu_after, 0);
    hipEventSynchronize(cu_after);
    c_after = clock();
    pcm_after = getSystemCounterState();
    GetProcessMemoryInfo(GetCurrentProcess(), &ps_after, sizeof(ps_after));
    result_filler(result, "Device to Host", c_before, c_after, pcm_before, pcm_after, ps_before, ps_after, cu_before, cu_after);

    //std::cout << "c[" << N * M - 1 << "]: " << h_c[1] << std::endl;

    // Cleanup after kernel execution
    hipFree(d_data);
    vccfree(vc);

    return result;
    
}
Json::Value malloc_cuda_copy(SIZE_T param_num) {

    SIZE_T allocSize = pow(2, 27);
    SIZE_T N = sqrt((allocSize / param_num)) / sqrt(sizeof(double));
    SIZE_T M = ((allocSize / param_num) / (N * sizeof(double)));

    Json::Value result;
    result["allocator"] = "m";
    result["scenario"] = "CUDA";
    result["datatype"] = typeid(double).name();
    result["size"] = param_num;

    double** h_data = (double**)malloc(sizeof(double*) * param_num);
    double** d_data = (double**)malloc(sizeof(double*) * param_num);


    //evaluation variables
    clock_t c_before, c_after;						//clock
    SystemCounterState pcm_before, pcm_after;		//intel pcm
    PROCESS_MEMORY_COUNTERS ps_before, ps_after;	//windows process status
    hipEvent_t cu_before, cu_after;
    hipEventCreate(&cu_before);
    hipEventCreate(&cu_after);



    //Host allocation
    for (size_t i = 0; i < param_num; i++)
        h_data[i] = (double*)malloc(sizeof(double) * N * M);

    // Host initialization
    for (SIZE_T i = 0; i < param_num; i++)
        for (size_t j = 0; j < N * M; j++)
            h_data[i][j] = i * N * M + j;

    // Device allocation
    for (size_t i = 0; i < param_num; i++) {
        hipMalloc((void**)&d_data[i], sizeof(double) * N * M);
        cuda_check();
    }

    // Transfer data from host to device memory
    //before
    GetProcessMemoryInfo(GetCurrentProcess(), &ps_before, sizeof(ps_before));
    pcm_before = getSystemCounterState();
    c_before = clock();
    hipEventRecord(cu_before, 0);

    for (size_t i = 0; i < param_num; i++){
       hipMemcpy(d_data[i], h_data[i], sizeof(double) * N * M, hipMemcpyHostToDevice);
       cuda_check();
    }
    //after
    hipEventRecord(cu_after, 0);
    hipEventSynchronize(cu_after);
    c_after = clock();
    pcm_after = getSystemCounterState();
    GetProcessMemoryInfo(GetCurrentProcess(), &ps_after, sizeof(ps_after));
    result_filler(result, "Host to Device", c_before, c_after, pcm_before, pcm_after, ps_before, ps_after, cu_before, cu_after);
    
    //Launch the calculation
    //vecAdd<<<M*N, 1>>> (&d_data[0], &d_data[N*M], &d_data[N*M*2]);
    //vecAdd_compact<<<M*N, 1>>> (d_data);
    //hipDeviceSynchronize();

    //Transfer data from device to host
    //before
    GetProcessMemoryInfo(GetCurrentProcess(), &ps_before, sizeof(ps_before));
    pcm_before = getSystemCounterState();
    c_before = clock();
    hipEventRecord(cu_before, 0);

    hipMemcpy(h_data[param_num-1], d_data[param_num - 1], sizeof(double) * N * M, hipMemcpyDeviceToHost);
    cuda_check();

    //after
    hipEventRecord(cu_after, 0);
    hipEventSynchronize(cu_after);
    c_after = clock();
    pcm_after = getSystemCounterState();
    GetProcessMemoryInfo(GetCurrentProcess(), &ps_after, sizeof(ps_after));
    result_filler(result, "Device to Host", c_before, c_after, pcm_before, pcm_after, ps_before, ps_after, cu_before, cu_after);

    //std::cout << "c[" << N * M - 1 << "]: " << h_c[1] << std::endl;

    // Cleanup after kernel execution
    for (size_t i = 0; i < param_num; i++)
    {
        hipFree(d_data[i]);
        free(h_data[i]);
    }

    return result;
    
}
Json::Value mi_malloc_cuda_copy(SIZE_T param_num) {

    SIZE_T allocSize = pow(2, 27);
    SIZE_T N = sqrt((allocSize / param_num)) / sqrt(sizeof(double));
    SIZE_T M = ((allocSize / param_num) / (N * sizeof(double)));

    Json::Value result;
    result["allocator"] = "mim";
    result["scenario"] = "CUDA";
    result["datatype"] = typeid(double).name();
    result["size"] = param_num;

    double** h_data = (double**)mi_malloc(sizeof(double*) * param_num);
    double** d_data = (double**)mi_malloc(sizeof(double*) * param_num);


    //evaluation variables
    clock_t c_before, c_after;						//clock
    SystemCounterState pcm_before, pcm_after;		//intel pcm
    PROCESS_MEMORY_COUNTERS ps_before, ps_after;	//windows process status
    hipEvent_t cu_before, cu_after;
    hipEventCreate(&cu_before);
    hipEventCreate(&cu_after);



    //Host allocation
    for (size_t i = 0; i < param_num; i++)
        h_data[i] = (double*)mi_malloc(sizeof(double) * N * M);

    // Host initialization
    for (SIZE_T i = 0; i < param_num; i++)
        for (size_t j = 0; j < N * M; j++)
            h_data[i][j] = i * N * M + j;

    // Device allocation
    for (size_t i = 0; i < param_num; i++) {
        hipMalloc((void**)&d_data[i], sizeof(double) * N * M);
        cuda_check();
    }

    // Transfer data from host to device memory
    //before
    GetProcessMemoryInfo(GetCurrentProcess(), &ps_before, sizeof(ps_before));
    pcm_before = getSystemCounterState();
    c_before = clock();
    hipEventRecord(cu_before, 0);

    for (size_t i = 0; i < param_num; i++) {
        hipMemcpy(d_data[i], h_data[i], sizeof(double) * N * M, hipMemcpyHostToDevice);
        cuda_check();
    }
    //after
    hipEventRecord(cu_after, 0);
    hipEventSynchronize(cu_after);
    c_after = clock();
    pcm_after = getSystemCounterState();
    GetProcessMemoryInfo(GetCurrentProcess(), &ps_after, sizeof(ps_after));
    result_filler(result, "Host to Device", c_before, c_after, pcm_before, pcm_after, ps_before, ps_after, cu_before, cu_after);

    //Launch the calculation
    //vecAdd<<<M*N, 1>>> (&d_data[0], &d_data[N*M], &d_data[N*M*2]);
    //vecAdd_compact<<<M*N, 1>>> (d_data);
    //hipDeviceSynchronize();

    //Transfer data from device to host
    //before
    GetProcessMemoryInfo(GetCurrentProcess(), &ps_before, sizeof(ps_before));
    pcm_before = getSystemCounterState();
    c_before = clock();
    hipEventRecord(cu_before, 0);

    hipMemcpy(h_data[param_num - 1], d_data[param_num - 1], sizeof(double) * N * M, hipMemcpyDeviceToHost);
    cuda_check();

    //after
    hipEventRecord(cu_after, 0);
    hipEventSynchronize(cu_after);
    c_after = clock();
    pcm_after = getSystemCounterState();
    GetProcessMemoryInfo(GetCurrentProcess(), &ps_after, sizeof(ps_after));
    result_filler(result, "Device to Host", c_before, c_after, pcm_before, pcm_after, ps_before, ps_after, cu_before, cu_after);

    //std::cout << "c[" << N * M - 1 << "]: " << h_c[1] << std::endl;

    // Cleanup after kernel execution
    for (size_t i = 0; i < param_num; i++)
    {
        hipFree(d_data[i]);
        mi_free(h_data[i]);
    }

    return result;

}


Json::Value vcmalloc_cuda_pin(SIZE_T param_num) {

    SIZE_T allocSize = pow(2, 27);
    SIZE_T N = sqrt((allocSize / param_num)) / sqrt(sizeof(double));
    SIZE_T M = ((allocSize / param_num) / (N * sizeof(double)));

    Json::Value result;
    result["allocator"] = "vcm";
    result["scenario"] = "CUDA";
    result["datatype"] = typeid(double).name();
    result["size"] = param_num;



    //evaluation variables
    clock_t c_before, c_after;						//clock
    SystemCounterState pcm_before, pcm_after;		//intel pcm
    PROCESS_MEMORY_COUNTERS ps_before, ps_after;	//windows process status
    hipEvent_t cu_before, cu_after;
    hipEventCreate(&cu_before);
    hipEventCreate(&cu_after);


    double** h_data;
    double* d_data;
    
    void* vc = vccalloc((SIZE_T)1024 * 1024 * 1024 * 1024);
    vcmalloc(vc, h_data, param_num);

    //Host allocation
    for (size_t i = 0; i < param_num; i++)
        vcmalloc(vc, h_data[i], N * M);

    // Host initialization
    for (SIZE_T i = 0; i < param_num; i++)
        for (size_t j = 0; j < N * M; j++)
            h_data[i][j] = i * N * M + j;

    // Device allocation
    hipMalloc((void**)&d_data, sizeof(double) * N * M * param_num);
    cuda_check();


    // Memory Pinning
    //before
    GetProcessMemoryInfo(GetCurrentProcess(), &ps_before, sizeof(ps_before));
    pcm_before = getSystemCounterState();
    c_before = clock();
    hipEventRecord(cu_before, 0);

    hipHostRegister(*h_data, sizeof(double) * N * M * param_num, hipHostRegisterPortable);
    cuda_check();

    //after
    hipEventRecord(cu_after, 0);
    hipEventSynchronize(cu_after);
    c_after = clock();
    pcm_after = getSystemCounterState();
    GetProcessMemoryInfo(GetCurrentProcess(), &ps_after, sizeof(ps_after));
    result_filler(result, "Pinning", c_before, c_after, pcm_before, pcm_after, ps_before, ps_after, cu_before, cu_after);

    // Transfer data from host to device memory
    //before
    GetProcessMemoryInfo(GetCurrentProcess(), &ps_before, sizeof(ps_before));
    pcm_before = getSystemCounterState();
    c_before = clock();
    hipEventRecord(cu_before, 0);

    hipMemcpy(d_data, *h_data, sizeof(double) * N * M * param_num, hipMemcpyHostToDevice);
    cuda_check();

    //after
    hipEventRecord(cu_after, 0);
    hipEventSynchronize(cu_after);
    c_after = clock();
    pcm_after = getSystemCounterState();
    GetProcessMemoryInfo(GetCurrentProcess(), &ps_after, sizeof(ps_after));
    result_filler(result, "Host to Device (Pinned)", c_before, c_after, pcm_before, pcm_after, ps_before, ps_after, cu_before, cu_after);

    // Launch the calculation
    // vecAdd<<<M*N, 1>>> (&d_data[0], &d_data[N*M], &d_data[N*M*2]);
    // vecAdd_compact<<<M*N, 1>>> (d_data);
    //hipDeviceSynchronize();

    //Transfer data from device to host
    //before
    GetProcessMemoryInfo(GetCurrentProcess(), &ps_before, sizeof(ps_before));
    pcm_before = getSystemCounterState();
    c_before = clock();
    hipEventRecord(cu_before, 0);

    hipMemcpy(h_data[param_num - 1], &d_data[M * N * (param_num - 1)], sizeof(double) * N * M, hipMemcpyDeviceToHost);
    cuda_check();

    //after
    hipEventRecord(cu_after, 0);
    hipEventSynchronize(cu_after);
    c_after = clock();
    pcm_after = getSystemCounterState();
    GetProcessMemoryInfo(GetCurrentProcess(), &ps_after, sizeof(ps_after));
    result_filler(result, "Device to Host (Pinned)", c_before, c_after, pcm_before, pcm_after, ps_before, ps_after, cu_before, cu_after);

    //std::cout << "c[" << N * M - 1 << "]: " << h_c[1] << std::endl;

    // Cleanup after kernel execution
    hipFree(d_data);
    vccfree(vc);

    return result;

}
Json::Value malloc_cuda_pin(SIZE_T param_num) {

    SIZE_T allocSize = pow(2, 27);
    SIZE_T N = sqrt((allocSize / param_num)) / sqrt(sizeof(double));
    SIZE_T M = ((allocSize / param_num) / (N * sizeof(double)));

    Json::Value result;
    result["allocator"] = "m";
    result["scenario"] = "CUDA";
    result["datatype"] = typeid(double).name();
    result["size"] = param_num;

    double** h_data = (double**)malloc(sizeof(double*) * param_num);
    double** d_data = (double**)malloc(sizeof(double*) * param_num);


    //evaluation variables
    clock_t c_before, c_after;						//clock
    SystemCounterState pcm_before, pcm_after;		//intel pcm
    PROCESS_MEMORY_COUNTERS ps_before, ps_after;	//windows process status
    hipEvent_t cu_before, cu_after;
    hipEventCreate(&cu_before);
    hipEventCreate(&cu_after);



    //Host allocation
    for (size_t i = 0; i < param_num; i++)
        h_data[i] = (double*)malloc(sizeof(double) * N * M);

    // Host initialization
    for (SIZE_T i = 0; i < param_num; i++)
        for (size_t j = 0; j < N * M; j++)
            h_data[i][j] = i * N * M + j;

    // Device allocation
    for (size_t i = 0; i < param_num; i++) {
        hipMalloc((void**)&d_data[i], sizeof(double) * N * M);
        cuda_check();
    }


    // Memory Pinning
    //before
    GetProcessMemoryInfo(GetCurrentProcess(), &ps_before, sizeof(ps_before));
    pcm_before = getSystemCounterState();
    c_before = clock();
    hipEventRecord(cu_before, 0);

    for (size_t i = 0; i < param_num; i++) {
        hipHostRegister(h_data[i], sizeof(double) * N * M, hipHostRegisterPortable);
        cuda_check();
    }
    //after
    hipEventRecord(cu_after, 0);
    hipEventSynchronize(cu_after);
    c_after = clock();
    pcm_after = getSystemCounterState();
    GetProcessMemoryInfo(GetCurrentProcess(), &ps_after, sizeof(ps_after));
    result_filler(result, "Pinning", c_before, c_after, pcm_before, pcm_after, ps_before, ps_after, cu_before, cu_after);



    // Transfer data from host to device memory
    //before
    GetProcessMemoryInfo(GetCurrentProcess(), &ps_before, sizeof(ps_before));
    pcm_before = getSystemCounterState();
    c_before = clock();
    hipEventRecord(cu_before, 0);

    for (size_t i = 0; i < param_num; i++) {
        hipMemcpy(d_data[i], h_data[i], sizeof(double) * N * M, hipMemcpyHostToDevice);
        cuda_check();
    }
    //after
    hipEventRecord(cu_after, 0);
    hipEventSynchronize(cu_after);
    c_after = clock();
    pcm_after = getSystemCounterState();
    GetProcessMemoryInfo(GetCurrentProcess(), &ps_after, sizeof(ps_after));
    result_filler(result, "Host to Device (Pinned)", c_before, c_after, pcm_before, pcm_after, ps_before, ps_after, cu_before, cu_after);

    // Launch the calculation
    // vecAdd<<<M*N, 1>>> (&d_data[0], &d_data[N*M], &d_data[N*M*2]);
    // vecAdd_compact<<<M*N, 1>>> (d_data);
    //hipDeviceSynchronize();

    //Transfer data from device to host
    //before
    GetProcessMemoryInfo(GetCurrentProcess(), &ps_before, sizeof(ps_before));
    pcm_before = getSystemCounterState();
    c_before = clock();
    hipEventRecord(cu_before, 0);

    hipMemcpy(h_data[param_num - 1], d_data[param_num - 1], sizeof(double) * N * M, hipMemcpyDeviceToHost);
    cuda_check();

    //after
    hipEventRecord(cu_after, 0);
    hipEventSynchronize(cu_after);
    c_after = clock();
    pcm_after = getSystemCounterState();
    GetProcessMemoryInfo(GetCurrentProcess(), &ps_after, sizeof(ps_after));
    result_filler(result, "Device to Host (Pinned)", c_before, c_after, pcm_before, pcm_after, ps_before, ps_after, cu_before, cu_after);

    //std::cout << "c[" << N * M - 1 << "]: " << h_c[1] << std::endl;

    // Cleanup after kernel execution
    for (size_t i = 0; i < param_num; i++)
    {
        hipFree(d_data[i]);
        free(h_data[i]);
    }

    return result;

}
Json::Value mi_malloc_cuda_pin(SIZE_T param_num) {

    SIZE_T allocSize = pow(2, 27);
    SIZE_T N = sqrt((allocSize / param_num)) / sqrt(sizeof(double));
    SIZE_T M = ((allocSize / param_num) / (N * sizeof(double)));

    Json::Value result;
    result["allocator"] = "mim";
    result["scenario"] = "CUDA";
    result["datatype"] = typeid(double).name();
    result["size"] = param_num;

    double** h_data = (double**)mi_malloc(sizeof(double*) * param_num);
    double** d_data = (double**)mi_malloc(sizeof(double*) * param_num);


    //evaluation variables
    clock_t c_before, c_after;						//clock
    SystemCounterState pcm_before, pcm_after;		//intel pcm
    PROCESS_MEMORY_COUNTERS ps_before, ps_after;	//windows process status
    hipEvent_t cu_before, cu_after;
    hipEventCreate(&cu_before);
    hipEventCreate(&cu_after);



    //Host allocation
    for (size_t i = 0; i < param_num; i++)
        h_data[i] = (double*)mi_malloc(sizeof(double) * N * M);

    // Host initialization
    for (SIZE_T i = 0; i < param_num; i++)
        for (size_t j = 0; j < N * M; j++)
            h_data[i][j] = i * N * M + j;

    // Device allocation
    for (size_t i = 0; i < param_num; i++) {
        hipMalloc((void**)&d_data[i], sizeof(double) * N * M);
        cuda_check();
    }


    // Memory Pinning
    //before
    GetProcessMemoryInfo(GetCurrentProcess(), &ps_before, sizeof(ps_before));
    pcm_before = getSystemCounterState();
    c_before = clock();
    hipEventRecord(cu_before, 0);

    for (size_t i = 0; i < param_num; i++) {
        hipHostRegister(h_data[i], sizeof(double) * N * M, hipHostRegisterPortable);
        cuda_check();
    }
    //after
    hipEventRecord(cu_after, 0);
    hipEventSynchronize(cu_after);
    c_after = clock();
    pcm_after = getSystemCounterState();
    GetProcessMemoryInfo(GetCurrentProcess(), &ps_after, sizeof(ps_after));
    result_filler(result, "Pinning", c_before, c_after, pcm_before, pcm_after, ps_before, ps_after, cu_before, cu_after);



    // Transfer data from host to device memory
    //before
    GetProcessMemoryInfo(GetCurrentProcess(), &ps_before, sizeof(ps_before));
    pcm_before = getSystemCounterState();
    c_before = clock();
    hipEventRecord(cu_before, 0);

    for (size_t i = 0; i < param_num; i++) {
        hipMemcpy(d_data[i], h_data[i], sizeof(double) * N * M, hipMemcpyHostToDevice);
        cuda_check();
    }
    //after
    hipEventRecord(cu_after, 0);
    hipEventSynchronize(cu_after);
    c_after = clock();
    pcm_after = getSystemCounterState();
    GetProcessMemoryInfo(GetCurrentProcess(), &ps_after, sizeof(ps_after));
    result_filler(result, "Host to Device (Pinned)", c_before, c_after, pcm_before, pcm_after, ps_before, ps_after, cu_before, cu_after);

    // Launch the calculation
    // vecAdd<<<M*N, 1>>> (&d_data[0], &d_data[N*M], &d_data[N*M*2]);
    // vecAdd_compact<<<M*N, 1>>> (d_data);
    //hipDeviceSynchronize();

    //Transfer data from device to host
    //before
    GetProcessMemoryInfo(GetCurrentProcess(), &ps_before, sizeof(ps_before));
    pcm_before = getSystemCounterState();
    c_before = clock();
    hipEventRecord(cu_before, 0);

    hipMemcpy(h_data[param_num - 1], d_data[param_num - 1], sizeof(double) * N * M, hipMemcpyDeviceToHost);
    cuda_check();

    //after
    hipEventRecord(cu_after, 0);
    hipEventSynchronize(cu_after);
    c_after = clock();
    pcm_after = getSystemCounterState();
    GetProcessMemoryInfo(GetCurrentProcess(), &ps_after, sizeof(ps_after));
    result_filler(result, "Device to Host (Pinned)", c_before, c_after, pcm_before, pcm_after, ps_before, ps_after, cu_before, cu_after);

    //std::cout << "c[" << N * M - 1 << "]: " << h_c[1] << std::endl;

    // Cleanup after kernel execution
    for (size_t i = 0; i < param_num; i++)
    {
        hipFree(d_data[i]);
        mi_free(h_data[i]);
    }

    return result;

}


int main(int argc, char* argv[]) {

    PCM* m = PCM::getInstance();
    m->resetPMU();
    if (m->program() != PCM::Success) return 0;

    typedef Json::Value(*tests_f)(SIZE_T allocSize);
    tests_f tests[] = {
        /*0*/	&vcmalloc_cuda_copy,
        /*1*/	&malloc_cuda_copy,
        /*2*/	&mi_malloc_cuda_copy,
        /*3*/   &vcmalloc_cuda_pin,
        /*4*/   &malloc_cuda_pin,
        /*5*/   &mi_malloc_cuda_pin

    };

    int test_id = stoi(argv[1]);
    int size_choice = stoi(argv[2]);
    int nbr_of_test = stoi(argv[3]);
    //SIZE_T size = pow(2, size_choice);

    to_json(tests[test_id](size_choice), nbr_of_test);

    m->cleanup();

    return 0;
}
